
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void Allocate_Memory(float **h_a, float **h_b, float **d_a, int N) {
    size_t size = N*sizeof(float);
    hipError_t Error;
    // Host memory
    *h_a = (float*)malloc(size); 
    *h_b = (float*)malloc(size); 
    // Device memory 
    Error = hipMalloc((void**)d_a, size); 
    printf("CUDA error (malloc d_a) = %s\n", hipGetErrorString(Error));
}

void Free_Memory(float **h_a, float **h_b, float **d_a) {
    if (*h_a) free(*h_a);
    if (*h_b) free(*h_b);
    if (*d_a) hipFree(*d_a);
}

void Send_To_Device(float **h_a, float **d_a, int N) {
    // Size of data to send
    size_t size = N*sizeof(float);
    // Grab a error type
    hipError_t Error;

    // Send A to the GPU
    Error = hipMemcpy(*d_a, *h_a, size, hipMemcpyHostToDevice); 
    printf("CUDA error (memcpy h_a -> d_a) = %s\n", hipGetErrorString(Error));
}

void Get_From_Device(float **d_a, float **h_b, int N) {
    // Size of data to send
    size_t size = N*sizeof(float);
    // Grab a error type
    hipError_t Error;
    // Send d_a to the host variable h_bS
    Error = hipMemcpy(*h_b, *d_a, size, hipMemcpyDeviceToHost);
    printf("CUDA error (memcpy d_a -> h_b) = %s\n", hipGetErrorString(Error));
}