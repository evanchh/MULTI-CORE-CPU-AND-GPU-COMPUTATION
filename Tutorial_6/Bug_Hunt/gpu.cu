
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void Allocate_Memory(float **h_a, float **d_a, int N) {
	size_t size = N*sizeof(float);
	hipError_t Error;
	// Host memory
	*h_a = (float*)malloc(size); 
	// Device memory
	Error = hipMalloc((void**)d_a, size); 
    printf("CUDA error (malloc d_a) = %s\n", hipGetErrorString(Error));
}

void Free_Memory(float **h_a, float **d_a) {
	if (*h_a) free(*h_a);
	if (*d_a) hipFree(*d_a);
}