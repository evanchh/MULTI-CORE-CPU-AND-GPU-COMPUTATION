#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void Set_GPU_Device(int device) {
    hipSetDevice(device);
}

void Allocate_CSR_Memory(int **d_row_ptr, int **d_col_idx, float **d_val, int NROWS, int NNZ) {
    hipError_t Error;
    // Device memory
    Error = hipMalloc((void**)d_row_ptr, (NROWS + 1) * sizeof(int));
    printf("CUDA error (malloc d_row_ptr) = %s\n", hipGetErrorString(Error));
    Error = hipMalloc((void**)d_col_idx, NNZ * sizeof(int));
    printf("CUDA error (malloc d_col_idx) = %s\n", hipGetErrorString(Error));
    Error = hipMalloc((void**)d_val, NNZ * sizeof(float));
    printf("CUDA error (malloc d_val) = %s\n", hipGetErrorString(Error));
}

void Free_CSR_Memory(int **row, int **col, float **val,
                     int **row_ptr, int **col_idx, float **val_csr,
                     int **d_row_ptr, int **d_col_idx, float **d_val, 
                     float **d_result, float **d_vec, float **d_out, 
                     float **d_vecA, float **d_vecB, float **d_dot, 
                     float **d_b, float **d_x) {
    if (*row) free(*row);
    if (*col) free(*col);
    if (*val) free(*val);
    if (*row_ptr) free(*row_ptr);
    if (*col_idx) free(*col_idx);
    if (*val_csr) free(*val_csr);
    if (*d_row_ptr) hipFree(*d_row_ptr);
    if (*d_col_idx) hipFree(*d_col_idx);
    if (*d_val) hipFree(*d_val);
    if (*d_result) hipFree(*d_result);
    if (*d_vec) hipFree(*d_vec);
    if (*d_out) hipFree(*d_out);
    if (*d_vecA) hipFree(*d_vecA);
    if (*d_vecB) hipFree(*d_vecB);
    if (*d_dot) hipFree(*d_dot);
    if (*d_b) hipFree(*d_b);
    if (*d_x) hipFree(*d_x);
}

// === COO -> CSR 上傳到 GPU ===
void Send_CSR_To_Device(int **d_row_ptr, int *h_row_ptr,
                        int **d_col_idx, int *h_col_idx,
                        float **d_val, float *h_val,
                        int NROWS, int NNZ) {
    // Grab an error type
    hipError_t Error;

    Error = hipMemcpy(*d_row_ptr, h_row_ptr, (NROWS + 1) * sizeof(int), hipMemcpyHostToDevice);
    printf("CUDA error (memcpy h_row_ptr -> d_row_ptr) = %s\n", hipGetErrorString(Error));

    Error = hipMemcpy(*d_col_idx, h_col_idx, NNZ * sizeof(int), hipMemcpyHostToDevice);
    printf("CUDA error (memcpy h_col_idx -> d_col_idx) = %s\n", hipGetErrorString(Error));

    Error = hipMemcpy(*d_val, h_val, NNZ * sizeof(float), hipMemcpyHostToDevice);
    printf("CUDA error (memcpy h_val -> d_val) = %s\n", hipGetErrorString(Error));
}

void Get_CSR_From_Device(int *d_row_ptr, int *h_row_ptr, 
                         int *d_col_idx, int *h_col_idx,
                         float *d_val, float *h_val,
                         int NROWS, int NNZ) {
    hipError_t Error;

    Error = hipMemcpy(h_row_ptr, d_row_ptr, (NROWS + 1) * sizeof(int), hipMemcpyDeviceToHost);
    printf("CUDA error (memcpy d_row_ptr -> h_row_ptr) = %s\n", hipGetErrorString(Error));

    Error = hipMemcpy(h_col_idx, d_col_idx, NNZ * sizeof(int), hipMemcpyDeviceToHost);
    printf("CUDA error (memcpy d_col_idx -> h_col_idx) = %s\n", hipGetErrorString(Error));

    Error = hipMemcpy(h_val, d_val, NNZ * sizeof(float), hipMemcpyDeviceToHost);
    printf("CUDA error (memcpy d_val -> h_val) = %s\n", hipGetErrorString(Error));
}

// === Milestone 2: 向量 x 常數乘法 kernel 和呼叫函數 ===
__global__ void Vector_Multiply_Constant(float *out, float *in, float alpha, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out[idx] = alpha * in[idx];
    }
}

void Launch_Vector_Multiply_Constant(float *d_out, float *d_in, float alpha, int N) {
    int threadsPerBlock = 128;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    Vector_Multiply_Constant<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_in, alpha, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize(); // optional for debug
}

// === Milestone 3: 矩陣 × 向量 (current implementation target) ===
__global__ void SpMV_CSR_Kernel(int *row_ptr, int *col_idx, float *val,
                                 float *vec, float *out, int num_rows) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0.0f;
        int start = row_ptr[row];
        int end = row_ptr[row + 1];
        for (int i = start; i < end; i++) {
            dot += val[i] * vec[col_idx[i]];
        }
        out[row] = dot;
    }
}

void Launch_SpMV_CSR(int *d_row_ptr, int *d_col_idx, float *d_val,
                     float *d_vec, float *d_out, int num_rows) {
    int threadsPerBlock = 128;
    int blocksPerGrid = (num_rows + threadsPerBlock - 1) / threadsPerBlock;
    SpMV_CSR_Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_row_ptr, d_col_idx, d_val, d_vec, d_out, num_rows);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error (SpMV): %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
}

// === Milestone 4: 向量內積 (Dot Product) ===
__global__ void DotProduct_Kernel(float *a, float *b, float *result, float *z, int N) {
    __shared__ float cache[128];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;

    __syncthreads();
    // Reduction
    // int i = blockDim.x / 2;
    // while (i != 0) {
    //     if (cacheIndex < i)
    //         cache[cacheIndex] += cache[cacheIndex + i];
    //     __syncthreads();
    //     i /= 2;
    // }
    // if (cacheIndex == 0)
    //     atomicAdd(result, cache[0]); //I explained why atomic add was bad. Why did you decide to use atomic add here?

    for (int i = blockDim.x / 2; i > 0; i = i /2 ){
        if(cacheIndex < i){
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }
    if (cacheIndex == 0) 
        z[blockIdx.x] = cache[0];
}

__global__ void FinalReduction(float *z, float *result, int size) {
    __shared__ float cache[128];
    int tid = threadIdx.x;

    if (tid < size)
        cache[tid] = z[tid];
    else
        cache[tid] = 0.0f;

    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            cache[tid] += cache[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)
        result[0] = cache[0];
}


void Launch_Dot_Product(float *d_a, float *d_b, float *d_result, float *d_tmp, int N) {
    hipMemset(d_result, 0, sizeof(float));
    hipMalloc(&d_tmp, 128 * sizeof(float));  // 分配 block 數量大小的暫存空間
    DotProduct_Kernel<<<128, 128>>>(d_a, d_b, d_result, d_tmp, N);
    FinalReduction<<<1, 128>>>(d_tmp, d_result, 128);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error (Dot Product): %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
}

__global__ void Vector_AXPY(float *y, float *x, float alpha, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) y[idx] += alpha * x[idx];
}

void Launch_Vector_AXPY(float *d_y, float *d_x, float alpha, int N) {
    int threadsPerBlock = 128;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    Vector_AXPY<<<blocksPerGrid, threadsPerBlock>>>(d_y, d_x, alpha, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error (AXPY): %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize(); // optional for debug
}

// === Milestone 5: 共軛梯度法 CG Solver ===
void Launch_CG_Solver(int *d_row_ptr, int *d_col_idx, float *d_val,
                      float *d_b, float *d_x, int N, int max_iter, float tol) {
    float *d_r, *d_p, *d_Ap, *d_rr, *d_pAp, *d_tmp;
    hipMalloc(&d_r, N * sizeof(float));
    hipMalloc(&d_p, N * sizeof(float));
    hipMalloc(&d_Ap, N * sizeof(float));
    hipMalloc(&d_rr, sizeof(float));
    hipMalloc(&d_pAp, sizeof(float));
    hipMalloc(&d_tmp, 256 * sizeof(float));

    // r = b - A*x
    Launch_SpMV_CSR(d_row_ptr, d_col_idx, d_val, d_x, d_Ap, N);          // Ap = A*x
    hipMemcpy(d_r, d_b, N * sizeof(float), hipMemcpyDeviceToDevice);   // r = b
    // Launch_Vector_Multiply_Constant(d_Ap, d_Ap, -1.0f, N);               // Ap = -Ap
    Launch_Vector_AXPY(d_r, d_Ap, -1.0f, N);                             // r = r + (-Ap)
    hipMemcpy(d_p, d_r, N * sizeof(float), hipMemcpyDeviceToDevice);   // p = r

    float alpha, beta, r_old, r_new;
    Launch_Dot_Product(d_r, d_r, d_rr, d_tmp, N);
    hipMemcpy(&r_old, d_rr, sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < max_iter && r_old > tol * tol; ++i) {
        Launch_SpMV_CSR(d_row_ptr, d_col_idx, d_val, d_p, d_Ap, N);
        Launch_Dot_Product(d_p, d_Ap, d_pAp, d_tmp, N);
        float pAp;
        hipMemcpy(&pAp, d_pAp, sizeof(float), hipMemcpyDeviceToHost);

        alpha = r_old / pAp;

        Launch_Vector_AXPY(d_x, d_p, alpha, N);                  // x += alpha * p
        Launch_Vector_AXPY(d_r, d_Ap, -alpha, N);                 // r -= alpha * Ap

        Launch_Dot_Product(d_r, d_r, d_rr, d_tmp, N);                   // r_new = dot(r, r)
        hipMemcpy(&r_new, d_rr, sizeof(float), hipMemcpyDeviceToHost);

        if (r_new < tol * tol) break; // 收斂條件提前結束

        beta = r_new / r_old;
        Launch_Vector_Multiply_Constant(d_p, d_p, beta, N);  // p = β*p
        Launch_Vector_AXPY(d_p, d_r, 1.0f, N);               // p = β*p + r

        r_old = r_new;
    }

    hipFree(d_r); hipFree(d_p); hipFree(d_Ap);
    hipFree(d_rr); hipFree(d_pAp); hipFree(d_tmp);
}